#include "hip/hip_runtime.h"

#include <stdio.h>
void checkDeviceMemory(void) {
	size_t free, total;
	hipMemGetInfo(& free, &total);	
	printf("Device memroy (free/total) = %lld/%lld bytes\n", free, total);
}



int main(void)
{
	int* dDataPtr;
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024 * 1024 * 12);
	// device 메모리를 할당.
// 할당된 메모리 공간의 시작 주소가 dDataPtr 포인터 변수에 저장.
// 주의: dDataPtr이 가리키는 주소는 device 메모리상의 주소이므로 
// 호스트 코드에서 dDataPtr을 통해 device 메모리에 직접 접근할 수는 없다.

	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024 * 1024 * 12);
	//메모리 공간의 값을 초기화. 0으로 초기화.
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));


	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	hipFree(dDataPtr);
	// device 메모리 해제


}