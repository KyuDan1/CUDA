#include "hip/hip_runtime.h"

#include <stdio.h>
void checkDeviceMemory(void) {
	size_t free, total;
	hipMemGetInfo(& free, &total);	
	printf("Device memroy (free/total) = %lld/%lld bytes\n", free, total);
}



int main(void)
{
	int* dDataPtr;
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024 * 1024 * 12);
	// device �޸𸮸� �Ҵ�.
// �Ҵ�� �޸� ������ ���� �ּҰ� dDataPtr ������ ������ ����.
// ����: dDataPtr�� ����Ű�� �ּҴ� device �޸𸮻��� �ּ��̹Ƿ� 
// ȣ��Ʈ �ڵ忡�� dDataPtr�� ���� device �޸𸮿� ���� ������ ���� ����.

	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024 * 1024 * 12);
	//�޸� ������ ���� �ʱ�ȭ. 0���� �ʱ�ȭ.
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));


	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	hipFree(dDataPtr);
	// device �޸� ����


}