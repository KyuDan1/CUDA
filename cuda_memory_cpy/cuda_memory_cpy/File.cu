#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

//printData kernel
//�Է��� �������ε� �̰��� �迭�� ù��° ��� �ּ���.
__global__ void printData(int* _dDataPtr) {
	printf("%d", _dDataPtr[threadIdx.x]);
}

__global__ void setData(int* _dDataPtr) {
	_dDataPtr[threadIdx.x] = 2;
}



int main(void) {
	
	// Define in host(CPU)
	int data[10] = { 0 };
	for (int i = 0; i < 10; i++) data[i] = 1;

	int* dDataPtr;

	// int�� 10���� �� �޸𸮰����� gpu���� �Ҵ�.
	// �Ҵ��� �޸��� ���� �ּҴ� dDataPtr ������ ������ ����ȴ�.
	hipMalloc(&dDataPtr, sizeof(int) * 10);

	// �� ������ 0���� �ʱ�ȭ.
	hipMemset(dDataPtr, 0, sizeof(int) * 10);

	printf("Data in device: ");
	printData << <1, 10 >> > (dDataPtr);

	// data �迭�� ���� dDataPtr �迭�� ����. ������ Host���� Device��
	hipMemcpy(dDataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
	printf("\nHost ->Device: ");
	printData << <1, 10 >> > (dDataPtr);

	setData << <1, 10 >> > (dDataPtr);

	hipMemcpy(data, dDataPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
	printf("\nDevice -> Host: ");
	for (int i = 0; i < 10; i++) printf("%d", data[i]);

	hipFree(dDataPtr);

}